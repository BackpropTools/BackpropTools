#include "hip/hip_runtime.h"
#define LAYER_IN_C_OPERATIONS_CPU_MUX_INCLUDE_CUDA
#include <layer_in_c/operations/cpu_mux.h>
// -------------- added for cuda training ----------------
#include <layer_in_c/nn/optimizers/adam/operations_cuda.h>
// -------------------------------------------------------
#include <layer_in_c/nn/operations_cpu_mux.h>
#include <layer_in_c/nn_models/operations_cpu.h>
#include <layer_in_c/nn_models/persist.h>
namespace lic = layer_in_c;
#include "../parameters_ppo.h"
#ifdef LAYER_IN_C_BACKEND_ENABLE_MKL
#include <layer_in_c/rl/components/on_policy_runner/operations_cpu_mkl.h>
#else
#ifdef LAYER_IN_C_BACKEND_ENABLE_ACCELERATE
#include <layer_in_c/rl/components/on_policy_runner/operations_cpu_accelerate.h>
#else
#include <layer_in_c/rl/components/on_policy_runner/operations_cpu.h>
#endif
#endif
// -------------- added for cuda training ----------------
#include <layer_in_c/rl/components/on_policy_runner/operations_generic_extensions.h>
// -------------------------------------------------------
#include <layer_in_c/rl/algorithms/ppo/operations_generic.h>
// -------------- added for cuda training ----------------
#include <layer_in_c/rl/algorithms/ppo/operations_generic_extensions.h>
// -------------------------------------------------------
#include <layer_in_c/rl/utils/evaluation.h>

#include <gtest/gtest.h>
#include <highfive/H5File.hpp>


namespace parameters = parameters_0;

using LOGGER = lic::devices::logging::CPU_TENSORBOARD;

using DEV_SPEC_SUPER = lic::devices::cpu::Specification<lic::devices::math::CPU, lic::devices::random::CPU, LOGGER>;
using TI = typename lic::DEVICE_FACTORY<DEV_SPEC_SUPER>::index_t;
namespace execution_hints{
    struct HINTS: lic::rl::components::on_policy_runner::ExecutionHints<TI, 16>{};
}
struct DEV_SPEC: DEV_SPEC_SUPER{
    using EXECUTION_HINTS = execution_hints::HINTS;
};


using DEVICE = lic::DEVICE_FACTORY<DEV_SPEC>;
using DEVICE_GPU = lic::DEVICE_FACTORY_GPU<lic::devices::DefaultCUDASpecification>;
using T = float;
using TI = typename DEVICE::index_t;


constexpr TI NUM_RUNS = 100;
constexpr TI ACTOR_CHECKPOINT_INTERVAL = 100000;
constexpr bool ENABLE_EVALUATION = true;
constexpr TI NUM_EVALUATION_EPISODES = 10;
constexpr TI EVALUATION_INTERVAL = 100000;
constexpr bool ACTOR_ENABLE_CHECKPOINTS = true;
constexpr bool ACTOR_OVERWRITE_CHECKPOINTS = false;
const std::string ACTOR_CHECKPOINT_DIRECTORY = "checkpoints/ppo_ant";

// --------------- changed for cuda training -----------------
TEST(LAYER_IN_C_RL_ENVIRONMENTS_MUJOCO_ANT, TRAINING_PPO_CUDA){
    // -------------------------------------------------------
    for(TI run_i = 0; run_i < NUM_RUNS; ++run_i){
        using penv = parameters::environment<double, TI>;
        using prl = parameters::rl<T, TI, penv::ENVIRONMENT>;
        // -------------- added for cuda training ----------------
        using ON_POLICY_RUNNER_COLLECTION_EVALUATION_BUFFER_TYPE = lic::rl::components::on_policy_runner::CollectionEvaluationBuffer<prl::ON_POLICY_RUNNER_SPEC>;
        using PPO_TRAINING_HYBRID_BUFFER_TYPE = lic::rl::algorithms::ppo::TrainingBuffersHybrid<prl::PPO_SPEC>;
        // -------------------------------------------------------

        std::string run_name = "ppo_ant_non_adaptive_lr";
        {
            auto now = std::chrono::system_clock::now();
            auto local_time = std::chrono::system_clock::to_time_t(now);
            std::tm* tm = std::localtime(&local_time);

            std::ostringstream oss;
            oss << std::put_time(tm, "%FT%T%z");
            run_name = oss.str() + "_" + run_name;
        }

        DEVICE::SPEC::LOGGING logger;
        DEVICE device;
        // -------------- added for cuda training ----------------
        DEVICE_GPU device_gpu;
        // -------------------------------------------------------
        prl::ACTOR_OPTIMIZER actor_optimizer;
        prl::CRITIC_OPTIMIZER critic_optimizer;
        auto rng = lic::random::default_engine(DEVICE::SPEC::RANDOM(), 500 + run_i);
        auto evaluation_rng = lic::random::default_engine(DEVICE::SPEC::RANDOM(), 12);
        prl::PPO_TYPE ppo, ppo_gpu;
        prl::PPO_BUFFERS_TYPE ppo_buffers;
        prl::ON_POLICY_RUNNER_TYPE on_policy_runner;
        prl::ON_POLICY_RUNNER_DATASET_TYPE on_policy_runner_dataset;
        // -------------- added for cuda training ----------------
        ON_POLICY_RUNNER_COLLECTION_EVALUATION_BUFFER_TYPE on_policy_runner_collection_eval_buffer_gpu, on_policy_runner_collection_eval_buffer_cpu;
        PPO_TRAINING_HYBRID_BUFFER_TYPE ppo_training_hybrid_buffer_cpu, ppo_training_hybrid_buffer_gpu;
        lic::Matrix<lic::matrix::Specification<T, TI, decltype(on_policy_runner_dataset.data)::ROWS, prl::PPO_SPEC::ENVIRONMENT::OBSERVATION_DIM>> gae_all_observations;
        lic::Matrix<lic::matrix::Specification<T, TI, decltype(on_policy_runner_dataset.data)::ROWS, 1>> gae_all_values;
        // -------------------------------------------------------
        // -------------- replaced for cuda training ----------------
        prl::ACTOR_EVAL_BUFFERS actor_eval_buffers, actor_eval_buffers_gpu;
        // ----------------------------------------------------------
        prl::ACTOR_BUFFERS actor_buffers;
        prl::CRITIC_BUFFERS critic_buffers;
        prl::CRITIC_BUFFERS_GAE critic_buffers_gae;
        penv::ENVIRONMENT envs[prl::N_ENVIRONMENTS];
        penv::ENVIRONMENT evaluation_env;
        bool ui = false;
        TI next_checkpoint_id = 0;
        TI next_evaluation_id = 0;

        // -------------- replaced for cuda training ----------------
        lic::init(device_gpu);
        lic::malloc(device, ppo);
        // -------------------------------------------------------
        lic::malloc(device, ppo_buffers);
        lic::malloc(device, on_policy_runner_dataset);
        // -------------- added for cuda training ----------------
        lic::malloc(device, on_policy_runner_collection_eval_buffer_cpu);
        lic::malloc(device, ppo_training_hybrid_buffer_cpu);
        // -------------------------------------------------------
        lic::malloc(device, on_policy_runner);
        lic::malloc(device, actor_eval_buffers);
        // ------------- removed for cuda training ---------------
//        lic::malloc(device, actor_buffers);
//        lic::malloc(device, critic_buffers);
//        lic::malloc(device, critic_buffers_gae);
        // -------------------------------------------------------
        for(auto& env : envs){
            lic::malloc(device, env);
        }
        lic::malloc(device, evaluation_env);
        // -------------- added for cuda training ----------------
        lic::malloc(device_gpu, actor_buffers);
        lic::malloc(device_gpu, critic_buffers);
        lic::malloc(device_gpu, critic_buffers_gae);
        lic::malloc(device_gpu, ppo_gpu);
        lic::malloc(device_gpu, on_policy_runner_collection_eval_buffer_gpu);
        lic::malloc(device_gpu, ppo_training_hybrid_buffer_gpu);
        lic::malloc(device_gpu, actor_eval_buffers_gpu);
        lic::malloc(device_gpu, gae_all_observations);
        lic::malloc(device_gpu, gae_all_values);
        // -------------------------------------------------------

        lic::init(device, on_policy_runner, envs, rng);
        // -------------- replaced for cuda training ----------------
        lic::init(device, ppo, actor_optimizer, critic_optimizer, rng);
        lic::copy(device_gpu, device, ppo_gpu, ppo);
        // -------------------------------------------------------
        device.logger = &logger;
        lic::construct(device, device.logger, run_name);
        auto training_start = std::chrono::high_resolution_clock::now();
        if(prl::PPO_SPEC::PARAMETERS::NORMALIZE_OBSERVATIONS){
            for(TI observation_normalization_warmup_step_i = 0; observation_normalization_warmup_step_i < prl::OBSERVATION_NORMALIZATION_WARMUP_STEPS; observation_normalization_warmup_step_i++) {
                lic::collect(device, on_policy_runner_dataset, on_policy_runner, ppo.actor, actor_eval_buffers, rng);
                update(device, ppo.observation_normalizer, on_policy_runner_dataset.observations);
            }
            lic::init(device, on_policy_runner, envs, rng); // reinitializing the on_policy_runner to reset the episode counters
        }
        for(TI ppo_step_i = 0; ppo_step_i < 2500; ppo_step_i++) {
            // -------------- added for cuda training ----------------
            lic::copy(device, device_gpu, ppo, ppo_gpu);
            // -------------------------------------------------------
            if(ACTOR_ENABLE_CHECKPOINTS && (on_policy_runner.step / ACTOR_CHECKPOINT_INTERVAL == next_checkpoint_id)){
                std::filesystem::path actor_output_dir = std::filesystem::path(ACTOR_CHECKPOINT_DIRECTORY) / run_name;
                try {
                    std::filesystem::create_directories(actor_output_dir);
                }
                catch (std::exception& e) {
                }
                std::string checkpoint_name = "latest.h5";
                if(!ACTOR_OVERWRITE_CHECKPOINTS){
                    std::stringstream checkpoint_name_ss;
                    checkpoint_name_ss << "actor_" << std::setw(15) << std::setfill('0') << next_checkpoint_id << "_" << std::setw(15) << std::setfill('0') << on_policy_runner.step << ".h5";
                    checkpoint_name = checkpoint_name_ss.str();
                }
                std::filesystem::path actor_output_path = actor_output_dir / checkpoint_name;
                try{
                    auto actor_file = HighFive::File(actor_output_path, HighFive::File::Overwrite);
                    lic::save(device, ppo.actor, actor_file.createGroup("actor"));
                }
                catch(HighFive::Exception& e){
                    std::cout << "Error while saving actor: " << e.what() << std::endl;
                }
                next_checkpoint_id++;
            }
            if(ENABLE_EVALUATION && (on_policy_runner.step / EVALUATION_INTERVAL == next_evaluation_id)){
                auto result = lic::evaluate(device, evaluation_env, ui, ppo.actor, lic::rl::utils::evaluation::Specification<NUM_EVALUATION_EPISODES, prl::ON_POLICY_RUNNER_STEP_LIMIT>(), evaluation_rng);
                lic::add_scalar(device, device.logger, "evaluation/return/mean", result.mean);
                lic::add_scalar(device, device.logger, "evaluation/return/std", result.std);
                lic::add_histogram(device, device.logger, "evaluation/return", result.returns, decltype(result)::N_EPISODES);
                std::cout << "Evaluation return mean: " << result.mean << " (std: " << result.std << ")" << std::endl;

//            if(step_i > 250000){
//                ASSERT_GT(mean_return, 1000);
//            }
                next_evaluation_id++;
            }
            device.logger->step = on_policy_runner.step;

            if(ppo_step_i % 1 == 0){
                std::chrono::duration<T> training_elapsed = std::chrono::high_resolution_clock::now() - training_start;
                std::cout << "PPO step: " << ppo_step_i << " elapsed: " << training_elapsed.count() << "s" << std::endl;
                lic::add_scalar(device, device.logger, "ppo/step", ppo_step_i);
                lic::add_scalar(device, device.logger, "ppo/actor_learning_rate", actor_optimizer.alpha);
                lic::add_scalar(device, device.logger, "ppo/critic_learning_rate", critic_optimizer.alpha);
            }
            for (TI action_i = 0; action_i < penv::ENVIRONMENT::ACTION_DIM; action_i++) {
                T action_log_std = lic::get(ppo.actor.log_std.parameters, 0, action_i);
                std::stringstream topic;
                topic << "actor/action_std/" << action_i;
                lic::add_scalar(device, device.logger, topic.str(), lic::math::exp(DEVICE::SPEC::MATH(), action_log_std));
            }
            auto start = std::chrono::high_resolution_clock::now();
            {
                auto start = std::chrono::high_resolution_clock::now();
                // -------------- replaced for cuda training ----------------
                lic::collect_hybrid(device, device_gpu, on_policy_runner_dataset, on_policy_runner, ppo.actor, ppo_gpu.actor, actor_eval_buffers_gpu, on_policy_runner_collection_eval_buffer_cpu, on_policy_runner_collection_eval_buffer_gpu, rng);
                // ----------------------------------------------------------
                if(prl::PPO_SPEC::PARAMETERS::NORMALIZE_OBSERVATIONS){
                    update(device, ppo.observation_normalizer, on_policy_runner_dataset.observations);
                    normalize(device, ppo.observation_normalizer, on_policy_runner_dataset.observations);
                }
                lic::add_scalar(device, device.logger, "opr/observation/mean", lic::mean(device, on_policy_runner_dataset.observations));
                lic::add_scalar(device, device.logger, "opr/observation/std", lic::std(device, on_policy_runner_dataset.observations));
                lic::add_scalar(device, device.logger, "opr/action/mean", lic::mean(device, on_policy_runner_dataset.actions));
                lic::add_scalar(device, device.logger, "opr/action/std", lic::std(device, on_policy_runner_dataset.actions));
                lic::add_scalar(device, device.logger, "opr/rewards/mean", lic::mean(device, on_policy_runner_dataset.rewards));
                lic::add_scalar(device, device.logger, "opr/rewards/std", lic::std(device, on_policy_runner_dataset.rewards));
                auto end = std::chrono::high_resolution_clock::now();
                std::chrono::duration<T> elapsed = end - start;
                std::cout << "Rollout: " << elapsed.count() << " s" << std::endl;
            }
            {
                auto start = std::chrono::high_resolution_clock::now();
                // -------------- replaced for cuda training ----------------
                copy(device_gpu, device, gae_all_observations, on_policy_runner_dataset.all_observations);
                evaluate(device_gpu, ppo_gpu.critic, gae_all_observations, gae_all_values, critic_buffers_gae);
                copy(device, device_gpu, on_policy_runner_dataset.all_values, gae_all_values);
                // ----------------------------------------------------------
                lic::estimate_generalized_advantages(device, on_policy_runner_dataset, prl::PPO_TYPE::SPEC::PARAMETERS{});
                auto end = std::chrono::high_resolution_clock::now();
                std::chrono::duration<T> elapsed = end - start;
                std::cout << "GAE: " << elapsed.count() << " s" << std::endl;
            }
            {
                auto start = std::chrono::high_resolution_clock::now();
                // -------------- replaced for cuda training ----------------
                lic::train_hybrid(device, device_gpu, ppo, ppo_gpu, on_policy_runner_dataset, actor_optimizer, critic_optimizer, ppo_buffers, ppo_training_hybrid_buffer_gpu, actor_buffers, critic_buffers, rng);
                // ----------------------------------------------------------
                auto end = std::chrono::high_resolution_clock::now();
                std::chrono::duration<T> elapsed = end - start;
                std::cout << "Train: " << elapsed.count() << " s" << std::endl;
            }
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<T> elapsed = end - start;
            std::cout << "Total: " << elapsed.count() << " s" << std::endl;
        }

        // -------------- replaced for cuda training ----------------
        lic::free(device, ppo);
        // -------------------------------------------------------
        lic::free(device, ppo_buffers);
        lic::free(device, on_policy_runner_dataset);
        // -------------- added for cuda training ----------------
        lic::free(device, on_policy_runner_collection_eval_buffer_cpu);
        lic::free(device, ppo_training_hybrid_buffer_cpu);
        // -------------------------------------------------------
        lic::free(device, on_policy_runner);
        lic::free(device, actor_eval_buffers);
        // ------------- removed for cuda training ---------------
//        lic::free(device, actor_buffers);
//        lic::free(device, critic_buffers);
//        lic::free(device, critic_buffers_gae);
        // -------------------------------------------------------
        for(auto& env : envs){
            lic::free(device, env);
        }
        lic::free(device, evaluation_env);
        // -------------- added for cuda training ----------------
        lic::free(device_gpu, actor_buffers);
        lic::free(device_gpu, critic_buffers);
        lic::free(device_gpu, critic_buffers_gae);
        lic::free(device_gpu, ppo_gpu);
        lic::free(device_gpu, on_policy_runner_collection_eval_buffer_gpu);
        lic::free(device_gpu, ppo_training_hybrid_buffer_gpu);
        lic::free(device_gpu, actor_eval_buffers_gpu);
        lic::free(device_gpu, gae_all_observations);
        lic::free(device_gpu, gae_all_values);
        // -------------------------------------------------------
    }

}
