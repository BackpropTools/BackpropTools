#define FUNCTION_PLACEMENT __device__ __host__

#include <layer_in_c/operations/hip/hip_runtime.h>
#include <layer_in_c/operations/cpu.h>

#include <layer_in_c/nn/operations_cuda.h>
#include <layer_in_c/nn_models/operations_cuda.h>
#include <layer_in_c/nn_models/operations_cpu.h>


#include "../../utils/utils.h"

//#include <gtest/gtest.h>

#include <random>
#include <chrono>
#include <highfive/H5File.hpp>

namespace lic = layer_in_c;

using DTYPE = double;


using DEVICE_CUDA = lic::devices::DefaultCUDA;
using DEVICE_CUDA_GENERIC = lic::devices::CUDA_GENERIC<DEVICE_CUDA::SPEC>;
using DEVICE_CPU = lic::devices::DefaultCPU;

template <typename DEVICE, typename T_T>
using StructureSpecification = lic::nn_models::mlp::StructureSpecification<T_T, typename DEVICE::index_t, 10, 5, 3, 64, lic::nn::activation_functions::GELU, lic::nn::activation_functions::IDENTITY>;


using NETWORK_SPEC_CUDA = lic::nn_models::mlp::AdamSpecification<StructureSpecification<DEVICE_CUDA_GENERIC, DTYPE>, lic::nn::optimizers::adam::DefaultParametersTF<DTYPE>>;
using NetworkType_CUDA = lic::nn_models::mlp::NeuralNetworkAdam<NETWORK_SPEC_CUDA>;
using NETWORK_SPEC_CPU = lic::nn_models::mlp::AdamSpecification<StructureSpecification<DEVICE_CPU, DTYPE>, lic::nn::optimizers::adam::DefaultParametersTF<DTYPE>>;
using NetworkType_CPU = lic::nn_models::mlp::NeuralNetworkAdam<NETWORK_SPEC_CPU>;

DEVICE_CPU::SPEC::LOGGING logger_cpu;
DEVICE_CPU device_cpu(logger_cpu);
NetworkType_CPU network_cpu;

DEVICE_CUDA::SPEC::LOGGING logger_cuda;
DEVICE_CUDA device_cuda(logger_cuda);
NetworkType_CUDA network_cuda;

//TEST(LAYER_IN_C_NN_MLP_CUDA, FULL_TRAINING) {
int main(){

    lic::reset_optimizer_state(device_cpu, network_cpu);
    lic::zero_gradient(device_cpu, network_cpu);
    auto rng = lic::random::default_engine(DEVICE_CPU::SPEC::RANDOM());
    lic::init_weights(device_cpu, network_cpu, rng);

    lic::copy(network_cuda, network_cpu);

    DTYPE input_cpu[NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM];
    DTYPE output_cpu[NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM];
    DTYPE d_loss_d_output_cpu[NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM];
    DTYPE d_input_cpu[NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM];
    for(size_t i = 0; i < NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM; ++i) {
        input_cpu[i] = lic::random::uniform_real_distribution(DEVICE_CPU::SPEC::RANDOM(), -(DTYPE)1, (DTYPE)1, rng);
    }
    for(size_t i = 0; i < NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM; ++i) {
        output_cpu[i] = lic::random::uniform_real_distribution(DEVICE_CPU::SPEC::RANDOM(), -(DTYPE)1, (DTYPE)1, rng);
    }

    {
        hipDeviceSynchronize();
        auto start = std::chrono::high_resolution_clock::now();
        for(DEVICE_CPU::index_t i = 0; i < 1000; ++i) {
            lic::forward(device_cpu, network_cpu, input_cpu);
        }
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::cout << "Elapsed time CPU forward: " << elapsed_seconds.count() * 1000 * 1000 << "us" << std::endl;
    }
    lic::nn::loss_functions::d_mse_d_x<DEVICE_CPU, DTYPE, NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM, 1>(device_cpu, network_cpu.output_layer.output, output_cpu, d_loss_d_output_cpu);
    DTYPE loss_cpu = lic::nn::loss_functions::mse<DEVICE_CPU, DTYPE, NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM, 1>(device_cpu, network_cpu.output_layer.output, output_cpu);
    lic::backward(device_cpu, network_cpu, input_cpu, d_loss_d_output_cpu, d_input_cpu);

    // GPU part
    DEVICE_CUDA* device_cuda_gpu;
    hipMalloc(&device_cuda_gpu, sizeof(DEVICE_CUDA));
    hipMemcpy(device_cuda_gpu, &device_cuda, sizeof(DEVICE_CUDA), hipMemcpyHostToDevice);

    NetworkType_CUDA* network_cuda_device;
    hipMalloc(&network_cuda_device, sizeof(NetworkType_CUDA));
    hipMemcpy(network_cuda_device, &network_cuda, sizeof(NetworkType_CUDA), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    DTYPE* input_gpu;
    hipMalloc(&input_gpu, sizeof(DTYPE) * NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM);
    hipMemcpy(input_gpu, input_cpu, sizeof(input_gpu) * NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Test first layer
    DTYPE* output_first_layer_gpu;
    hipMalloc(&output_first_layer_gpu, sizeof(DTYPE) * NETWORK_SPEC_CPU::INPUT_LAYER::SPEC::OUTPUT_DIM);

    {
        hipDeviceSynchronize();
        auto start = std::chrono::high_resolution_clock::now();
        for(DEVICE_CPU::index_t i = 0; i < 1000; ++i) {
            lic::evaluate(*device_cuda_gpu, network_cuda_device->input_layer, input_gpu, output_first_layer_gpu);
        }
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::cout << "Elapsed time GPU layer: " << elapsed_seconds.count() * 1000 * 1000 << "us" << std::endl;
    }

    DTYPE output_first_layer_gpu_cpu[NETWORK_SPEC_CPU::INPUT_LAYER::SPEC::OUTPUT_DIM];
    hipMemcpy(output_first_layer_gpu_cpu, output_first_layer_gpu, sizeof(DTYPE) * NETWORK_SPEC_CPU::INPUT_LAYER::SPEC::OUTPUT_DIM, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    DTYPE output_first_layer_diff = lic::nn::layers::dense::helper::abs_diff_vector<DTYPE, NETWORK_SPEC_CPU::INPUT_LAYER::SPEC::OUTPUT_DIM>(output_first_layer_gpu_cpu, network_cpu.input_layer.output);

    std::cout << "CPU - CUDA evaluation diff input layer: " << output_first_layer_diff << std::endl;
//    assert(output_first_layer_diff < 1e-15);

    // Test full network
    DTYPE* output_full_network_gpu;
    hipMalloc(&output_full_network_gpu, sizeof(DTYPE) * NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM);

    {
        DTYPE* layer_output_tick;
        DTYPE* layer_output_tock;
        hipMalloc((void**)&layer_output_tick, sizeof(DTYPE) * NETWORK_SPEC_CUDA::STRUCTURE_SPEC::HIDDEN_DIM);
        hipMalloc((void**)&layer_output_tock, sizeof(DTYPE) * NETWORK_SPEC_CUDA::STRUCTURE_SPEC::HIDDEN_DIM);
        hipDeviceSynchronize();
        auto start = std::chrono::high_resolution_clock::now();
        for(DEVICE_CPU::index_t i = 0; i < 1000; ++i) {
            lic::evaluate_memless(*device_cuda_gpu, *network_cuda_device, input_gpu, output_full_network_gpu, layer_output_tick, layer_output_tock);
        }
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        hipFree((void**)&layer_output_tick);
        hipFree((void**)&layer_output_tock);
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::cout << "Elapsed time GPU forward: " << elapsed_seconds.count() * 1000 * 1000 << "us" << std::endl;
    }

    DTYPE output_full_network_gpu_cpu[NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM];
    hipMemcpy(output_full_network_gpu_cpu, output_full_network_gpu, sizeof(DTYPE) * NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    DTYPE output_full_network_diff = lic::nn::layers::dense::helper::abs_diff_vector<DTYPE, NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM>(output_full_network_gpu_cpu, network_cpu.output_layer.output);

    std::cout << "CPU - CUDA evaluation diff full output: " << output_full_network_diff << std::endl;
//    assert(output_full_network_diff < 1e-15);
    return 0;
}
