#define FUNCTION_PLACEMENT __device__ __host__

#include <layer_in_c/operations/hip/hip_runtime.h>
#include <layer_in_c/operations/cpu.h>

#include <layer_in_c/nn/operations_cuda.h>
#include <layer_in_c/nn_models/operations_cuda.h>
#include <layer_in_c/nn_models/operations_cpu.h>


#include "../../utils/utils.h"

//#include <gtest/gtest.h>

#include <random>
#include <chrono>
#include <highfive/H5File.hpp>

#ifdef LAYER_IN_C_TESTS_NN_CUDA_ENABLE_CUTLASS
#include "cutlass/gemm/device/gemm.h"
#endif

namespace lic = layer_in_c;

using DTYPE = float;


using DEVICE_CUDA = lic::devices::DefaultCUDA;
using DEVICE_CUDA_GENERIC = lic::devices::CUDA_GENERIC<DEVICE_CUDA::SPEC>;
using DEVICE_CPU = lic::devices::DefaultCPU;

constexpr DEVICE_CPU::index_t BATCH_SIZE = 1000;

template <typename DEVICE, typename T_T>
using StructureSpecification = lic::nn_models::mlp::StructureSpecification<T_T, typename DEVICE::index_t, 10, 5, 3, 64, lic::nn::activation_functions::RELU, lic::nn::activation_functions::IDENTITY>;


using NETWORK_SPEC_CUDA = lic::nn_models::mlp::AdamSpecification<StructureSpecification<DEVICE_CUDA_GENERIC, DTYPE>, lic::nn::optimizers::adam::DefaultParametersTF<DTYPE>>;
using NetworkType_CUDA = lic::nn_models::mlp::NeuralNetworkAdam<NETWORK_SPEC_CUDA>;
using NETWORK_SPEC_CPU = lic::nn_models::mlp::AdamSpecification<StructureSpecification<DEVICE_CPU, DTYPE>, lic::nn::optimizers::adam::DefaultParametersTF<DTYPE>>;
using NetworkType_CPU = lic::nn_models::mlp::NeuralNetworkAdam<NETWORK_SPEC_CPU>;

DEVICE_CPU::SPEC::LOGGING logger_cpu;
DEVICE_CPU device_cpu(logger_cpu);
NetworkType_CPU network_cpu;

DEVICE_CUDA::SPEC::LOGGING logger_cuda;
DEVICE_CUDA device_cuda(logger_cuda);
NetworkType_CUDA network_cuda;

//TEST(LAYER_IN_C_NN_MLP_CUDA, FULL_TRAINING) {
int main(){

    lic::reset_optimizer_state(device_cpu, network_cpu);
    lic::zero_gradient(device_cpu, network_cpu);
    auto rng = lic::random::default_engine(DEVICE_CPU::SPEC::RANDOM());
    lic::init_weights(device_cpu, network_cpu, rng);

    lic::copy(network_cuda, network_cpu);

    DTYPE input_cpu[BATCH_SIZE][NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM];
    DTYPE output_cpu[BATCH_SIZE][NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM];
    DTYPE output_first_layer_cpu[BATCH_SIZE][NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM];
    for(DEVICE_CPU::CPU::index_t batch_i = 0; batch_i < BATCH_SIZE; batch_i++){
        for(size_t i = 0; i < NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM; ++i) {
            input_cpu[batch_i][i] = lic::random::uniform_real_distribution(DEVICE_CPU::SPEC::RANDOM(), -(DTYPE) 1, (DTYPE) 1, rng);
        }
    }

    constexpr unsigned NUM_ITERATIONS = 1;

    {
        auto start = std::chrono::high_resolution_clock::now();
        for(DEVICE_CPU::index_t i = 0; i < NUM_ITERATIONS; ++i) {
            for(DEVICE_CPU::CPU::index_t batch_i = 0; batch_i < BATCH_SIZE; batch_i++){
                lic::forward(device_cpu, network_cpu, input_cpu[batch_i], output_cpu[batch_i]);
                memcpy(output_first_layer_cpu[batch_i], network_cpu.input_layer.output, sizeof(DTYPE) * NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM);
            }
        }
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::cout << "Elapsed time CPU forward: " << elapsed_seconds.count() * 1000 * 1000 << " us" << std::endl;
    }

    // GPU part
    DEVICE_CUDA* device_cuda_gpu;
    hipMalloc(&device_cuda_gpu, sizeof(DEVICE_CUDA));
    hipMemcpy(device_cuda_gpu, &device_cuda, sizeof(DEVICE_CUDA), hipMemcpyHostToDevice);

    NetworkType_CUDA* network_cuda_device;
    hipMalloc(&network_cuda_device, sizeof(NetworkType_CUDA));
    hipMemcpy(network_cuda_device, &network_cuda, sizeof(NetworkType_CUDA), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    DTYPE* input_gpu;
    hipMalloc(&input_gpu, sizeof(DTYPE) * BATCH_SIZE * NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM);
    hipMemcpy(input_gpu, input_cpu, sizeof(DTYPE) * BATCH_SIZE * NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Test first layer

    {
        DTYPE* output_first_layer_gpu;
        hipMalloc(&output_first_layer_gpu, sizeof(DTYPE) * BATCH_SIZE * NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM);
        hipDeviceSynchronize();
        auto start = std::chrono::high_resolution_clock::now();
        for(DEVICE_CPU::index_t i = 0; i < NUM_ITERATIONS; ++i) {
            for(DEVICE_CPU::CPU::index_t batch_i = 0; batch_i < BATCH_SIZE; batch_i++) {
                lic::evaluate(*device_cuda_gpu, network_cuda_device->input_layer, &input_gpu[batch_i * NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM], &output_first_layer_gpu[batch_i * NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM]);
            }
        }
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::cout << "Elapsed time GPU layer: " << elapsed_seconds.count() * 1000 * 1000 << " us" << std::endl;

        DTYPE output_first_layer_gpu_cpu[BATCH_SIZE][NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM];
        hipMemcpy(output_first_layer_gpu_cpu, output_first_layer_gpu, sizeof(DTYPE) * BATCH_SIZE * NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        DTYPE output_first_layer_diff_per_weight = lic::nn::layers::dense::helper::abs_diff_matrix<DTYPE, BATCH_SIZE, NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM>(output_first_layer_gpu_cpu, output_first_layer_cpu) / NetworkType_CUDA::NUM_WEIGHTS ;

        std::cout << "CPU - CUDA evaluation diff input layer: " << output_first_layer_diff_per_weight << std::endl;
        assert(output_first_layer_diff_per_weight < 1e-7);
    }





#ifdef LAYER_IN_C_TESTS_NN_CUDA_ENABLE_CUTLASS
    // Speed tests CUTLASS
    {
        constexpr unsigned M = NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM;
        constexpr unsigned K = NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM;
        constexpr unsigned N = 1;
        constexpr DTYPE alpha = 1, beta = 1;
        constexpr unsigned lda = M, ldb = K, ldc = M;
        using Majority = cutlass::layout::RowMajor;
        using CutlassGemm = cutlass::gemm::device::Gemm<DTYPE,        // Data-type of A matrix
                Majority,  // Layout of A matrix
                float,        // Data-type of B matrix
                Majority,  // Layout of B matrix
                float,        // Data-type of C matrix
                Majority>; // Layout of C matrix
        CutlassGemm::Arguments args({M, N, K},  // Gemm Problem dimensions
                                    {(DTYPE *) network_cuda_device->input_layer.weights, K},    // Tensor-ref for source matrix A
                                    {input_gpu, N},    // Tensor-ref for source matrix B
                                    {(DTYPE *) network_cuda_device->input_layer.biases, N},    // Tensor-ref for source matrix C
                                    {output_first_layer_gpu, N},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                                    {alpha, beta}); // Scalars used in the Epilogue

        CutlassGemm gemm_operator;
        hipDeviceSynchronize();
        auto start = std::chrono::high_resolution_clock::now();
        for(DEVICE_CPU::index_t i = 0; i < NUM_ITERATIONS; ++i) {
            cutlass::Status status = gemm_operator(args);
        }
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::cout << "Elapsed time CUTLASS layer: " << elapsed_seconds.count() * 1000 * 1000 << " us" << std::endl;
    }

    hipMemcpy(output_first_layer_gpu_cpu, output_first_layer_gpu, sizeof(DTYPE) * NETWORK_SPEC_CPU::INPUT_LAYER::SPEC::OUTPUT_DIM, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    DTYPE output_first_layer_cutlass_diff = lic::nn::layers::dense::helper::abs_diff_vector<DTYPE, NETWORK_SPEC_CPU::INPUT_LAYER::SPEC::OUTPUT_DIM>(output_first_layer_gpu_cpu, network_cpu.input_layer.output);

    std::cout << "CPU - CUDA evaluation diff input layer cutlass: " << output_first_layer_cutlass_diff << std::endl;
#endif

    // Test full network
    DTYPE* output_full_network_gpu;
    hipMalloc(&output_full_network_gpu, sizeof(DTYPE) * BATCH_SIZE * NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM);

    {
        DTYPE* layer_output_tick;
        DTYPE* layer_output_tock;
        hipMalloc((void**)&layer_output_tick, sizeof(DTYPE) * NETWORK_SPEC_CUDA::STRUCTURE_SPEC::HIDDEN_DIM);
        hipMalloc((void**)&layer_output_tock, sizeof(DTYPE) * NETWORK_SPEC_CUDA::STRUCTURE_SPEC::HIDDEN_DIM);
        hipDeviceSynchronize();
        auto start = std::chrono::high_resolution_clock::now();
        for(DEVICE_CPU::index_t i = 0; i < NUM_ITERATIONS; ++i) {
            for(DEVICE_CPU::CPU::index_t batch_i = 0; batch_i < BATCH_SIZE; batch_i++) {
                lic::evaluate_memless(*device_cuda_gpu, *network_cuda_device, &input_gpu[batch_i * NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM], &output_full_network_gpu[batch_i * NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM], layer_output_tick, layer_output_tock);
            }
        }
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
//        hipFree((void**)&layer_output_tick);
//        hipFree((void**)&layer_output_tock);
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::cout << "Elapsed time GPU forward: " << elapsed_seconds.count() * 1000 * 1000 << " us" << std::endl;
    }

    DTYPE output_full_network_gpu_cpu[BATCH_SIZE][NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM];
    hipMemcpy(output_full_network_gpu_cpu, output_full_network_gpu, sizeof(DTYPE) * BATCH_SIZE * NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    DTYPE output_full_network_diff = lic::nn::layers::dense::helper::abs_diff_matrix<DTYPE, BATCH_SIZE, NETWORK_SPEC_CPU::STRUCTURE_SPEC::OUTPUT_DIM>(output_full_network_gpu_cpu, output_cpu);

    std::cout << "CPU - CUDA evaluation diff full output: " << output_full_network_diff << std::endl;

    // Test batch layer evaluation
    {
        DTYPE* output;
        hipMalloc(&output, sizeof(DTYPE) * BATCH_SIZE * NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM);

        DTYPE* input;
        hipMalloc(&input, sizeof(DTYPE) * BATCH_SIZE * NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM);
        hipMemcpy(input, input_cpu, sizeof(DTYPE) * BATCH_SIZE * NETWORK_SPEC_CPU::STRUCTURE_SPEC::INPUT_DIM, hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        auto start = std::chrono::high_resolution_clock::now();
        for(DEVICE_CPU::index_t i = 0; i < NUM_ITERATIONS; ++i) {
            lic::evaluate_batch<DEVICE_CUDA::SPEC, NetworkType_CUDA::SPEC::INPUT_LAYER::SPEC, BATCH_SIZE>(*device_cuda_gpu, network_cuda_device->input_layer, input, output);
        }
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::cout << "Elapsed time GPU batch forward: " << elapsed_seconds.count() * 1000 * 1000 << " us" << std::endl;


        DTYPE output_layer_batch_gpu_cpu[BATCH_SIZE][NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM];
        hipMemcpy(output_layer_batch_gpu_cpu, output, sizeof(DTYPE) * BATCH_SIZE * NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        DTYPE output_layer_batch_network_diff_per_weight = lic::nn::layers::dense::helper::abs_diff_matrix<DTYPE, BATCH_SIZE, NETWORK_SPEC_CPU::STRUCTURE_SPEC::HIDDEN_DIM>(output_layer_batch_gpu_cpu, output_first_layer_cpu) / NetworkType_CUDA::NUM_WEIGHTS;
        std::cout << "CPU - CUDA evaluation batch diff: " << output_layer_batch_network_diff_per_weight << std::endl;
        auto layer_cpu = network_cpu.input_layer;
        decltype(layer_cpu)* input_layer_gpu;
        hipMalloc(&input_layer_gpu, sizeof(decltype(layer_cpu)));
        hipMemcpy(input_layer_gpu, &network_cuda.input_layer, sizeof(decltype(layer_cpu)), hipMemcpyDeviceToHost);
        assert(output_layer_batch_network_diff_per_weight < 1e-15);
    }

    return 0;
}
