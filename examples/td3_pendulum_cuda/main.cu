#include "hip/hip_runtime.h"
#include <layer_in_c/operations/cpu.h>


#define FUNCTION_PLACEMENT __device__ __host__

#include <layer_in_c/operations/hip/hip_runtime.h>

#include <layer_in_c/nn_models/operations_generic.h>
#include <layer_in_c/rl/environments/pendulum/operations_generic.h>
#include <layer_in_c/rl/components/off_policy_runner/operations_generic.h>
#include <layer_in_c/rl/algorithms/td3/operations_generic.h>

#include <layer_in_c/rl/utils/evaluation.h>

namespace lic = layer_in_c;
using DTYPE = float;

using DEVICE = lic::devices::DefaultCPU;
typedef lic::rl::environments::pendulum::Specification<DTYPE, lic::rl::environments::pendulum::DefaultParameters<DTYPE>> PENDULUM_SPEC;
typedef lic::rl::environments::Pendulum<DEVICE, PENDULUM_SPEC> ENVIRONMENT;
ENVIRONMENT env;

struct ActorStructureSpec{
    using T = DTYPE;
    static constexpr size_t INPUT_DIM = ENVIRONMENT::OBSERVATION_DIM;
    static constexpr size_t OUTPUT_DIM = ENVIRONMENT::ACTION_DIM;
    static constexpr int NUM_LAYERS = 3;
    static constexpr int HIDDEN_DIM = 64;
    static constexpr lic::nn::activation_functions::ActivationFunction HIDDEN_ACTIVATION_FUNCTION = lic::nn::activation_functions::RELU;
    static constexpr lic::nn::activation_functions::ActivationFunction OUTPUT_ACTIVATION_FUNCTION = lic::nn::activation_functions::TANH;
};

struct CriticStructureSpec{
    using T = DTYPE;
    static constexpr size_t INPUT_DIM = ENVIRONMENT::OBSERVATION_DIM + ENVIRONMENT::ACTION_DIM;
    static constexpr size_t OUTPUT_DIM = 1;
    static constexpr int NUM_LAYERS = 3;
    static constexpr int HIDDEN_DIM = 64;
    static constexpr lic::nn::activation_functions::ActivationFunction HIDDEN_ACTIVATION_FUNCTION = lic::nn::activation_functions::RELU;
    static constexpr lic::nn::activation_functions::ActivationFunction OUTPUT_ACTIVATION_FUNCTION = lic::nn::activation_functions::IDENTITY;
};

using AC_DEVICE = lic::devices::DefaultCPU;
using AC_DEVICE_CUDA = lic::devices::DefaultCUDA;
template <typename T>
struct TD3PendulumParameters: lic::rl::algorithms::td3::DefaultParameters<AC_DEVICE, T>{
    constexpr static size_t CRITIC_BATCH_SIZE = 100;
    constexpr static size_t ACTOR_BATCH_SIZE = 100;
};

using NN_DEVICE = lic::devices::DefaultCPU;
using NN_DEVICE_CUDA = lic::devices::DefaultCUDA;

template <typename DEVICE>
struct NetworkTypes{
    using ACTOR_NETWORK_SPEC = lic::nn_models::mlp::AdamSpecification<DEVICE, ActorStructureSpec, typename lic::nn::optimizers::adam::DefaultParametersTorch<DTYPE>>;
    using ACTOR_NETWORK_TYPE = lic::nn_models::mlp::NeuralNetworkAdam<DEVICE, ACTOR_NETWORK_SPEC>;

    using ACTOR_TARGET_NETWORK_SPEC = lic::nn_models::mlp::InferenceSpecification<DEVICE, ActorStructureSpec>;
    using ACTOR_TARGET_NETWORK_TYPE = layer_in_c::nn_models::mlp::NeuralNetwork<DEVICE , ACTOR_TARGET_NETWORK_SPEC>;

    using CRITIC_NETWORK_SPEC = lic::nn_models::mlp::AdamSpecification<DEVICE, CriticStructureSpec, typename lic::nn::optimizers::adam::DefaultParametersTorch<DTYPE>>;
    using CRITIC_NETWORK_TYPE = layer_in_c::nn_models::mlp::NeuralNetworkAdam<DEVICE, CRITIC_NETWORK_SPEC>;

    using CRITIC_TARGET_NETWORK_SPEC = layer_in_c::nn_models::mlp::InferenceSpecification<DEVICE, CriticStructureSpec>;
    using CRITIC_TARGET_NETWORK_TYPE = layer_in_c::nn_models::mlp::NeuralNetwork<DEVICE, CRITIC_TARGET_NETWORK_SPEC>;
};

using NetworkTypesCPU = NetworkTypes<NN_DEVICE>;
using NetworkTypesCUDA = NetworkTypes<NN_DEVICE_CUDA>;
using TD3_SPEC = lic::rl::algorithms::td3::Specification<DTYPE, ENVIRONMENT, NN_DEVICE, NetworkTypesCPU::ACTOR_NETWORK_TYPE, NetworkTypesCPU::ACTOR_TARGET_NETWORK_TYPE, NetworkTypesCPU::CRITIC_NETWORK_TYPE, NetworkTypesCPU::CRITIC_TARGET_NETWORK_TYPE, TD3PendulumParameters<DTYPE>>;
using TD3_SPEC_CUDA = lic::rl::algorithms::td3::Specification<DTYPE, ENVIRONMENT, NN_DEVICE, NetworkTypesCUDA::ACTOR_NETWORK_TYPE, NetworkTypesCUDA::ACTOR_TARGET_NETWORK_TYPE, NetworkTypesCUDA::CRITIC_NETWORK_TYPE, NetworkTypesCUDA::CRITIC_TARGET_NETWORK_TYPE, TD3PendulumParameters<DTYPE>>;
using ActorCriticType = lic::rl::algorithms::td3::ActorCritic<AC_DEVICE, TD3_SPEC>;
using ActorCriticTypeCUDA = lic::rl::algorithms::td3::ActorCritic<AC_DEVICE_CUDA, TD3_SPEC_CUDA>;



constexpr size_t REPLAY_BUFFER_CAP = 500000;
constexpr size_t ENVIRONMENT_STEP_LIMIT = 200;
AC_DEVICE::SPEC::LOGGING logger;
AC_DEVICE device(logger);
NN_DEVICE nn_device(logger);
lic::rl::components::OffPolicyRunner<
        AC_DEVICE,
        lic::rl::components::off_policy_runner::Specification<
                AC_DEVICE,
                DTYPE,
                ENVIRONMENT,
                REPLAY_BUFFER_CAP,
                ENVIRONMENT_STEP_LIMIT,
                lic::rl::components::off_policy_runner::DefaultParameters<DTYPE>
        >
> off_policy_runner(device);
ActorCriticType actor_critic(device, nn_device);
const DTYPE STATE_TOLERANCE = 0.00001;
constexpr int N_WARMUP_STEPS = ActorCriticType::SPEC::PARAMETERS::ACTOR_BATCH_SIZE;
static_assert(ActorCriticType::SPEC::PARAMETERS::ACTOR_BATCH_SIZE == ActorCriticType::SPEC::PARAMETERS::CRITIC_BATCH_SIZE);

#include <iostream>

template <typename C, typename RB, typename RNG>
__global__ void
train_critic_kernel(const ActorCriticTypeCUDA* pac, C* pc, const RB* prb, RNG* prng, bool init){
    if(blockIdx.x == 0 && threadIdx.x == 0){
        printf("train_critic_kernel\n");
        if(init){
            hiprand_init(0, 0, 0, prng);
        }
        ActorCriticTypeCUDA ac = *pac;
        C c = *pc;
        RB rb = *prb;
        RNG rng = *prng;
        lic::train_critic(ac, c, rb, rng);
        *prng = rng;
        lic::copy(pc, &c);
    }
}

int main() {
    std::mt19937 rng(2);
    lic::init(actor_critic, rng);
    ActorCriticTypeCUDA                            * actor_critic_gpu;
    ActorCriticTypeCUDA::SPEC::CRITIC_NETWORK_TYPE * critic_2_gpu;
    decltype(off_policy_runner.replay_buffer)      * rb_gpu;
    hiprandState* rng_gpu;
    hipMalloc(&actor_critic_gpu, sizeof(actor_critic));
    hipMalloc(&    critic_2_gpu, sizeof(actor_critic.critic_2));
    hipMalloc(&          rb_gpu, sizeof(off_policy_runner.replay_buffer));
    hipMalloc(&         rng_gpu, sizeof(hiprandState));
    hipDeviceSynchronize();

    for(int step_i = 0; step_i < 15000; step_i++){
        if(step_i > REPLAY_BUFFER_CAP){
            std::cout << "warning: replay buffer is rolling over" << std::endl;
        }
        lic::step(off_policy_runner, actor_critic.actor, rng);

        if(off_policy_runner.replay_buffer.full || off_policy_runner.replay_buffer.position > N_WARMUP_STEPS){
            if(step_i % 1000 == 0){
                std::cout << "step_i: " << step_i << std::endl;
            }
//            DTYPE critic_1_loss = lic::train_critic(actor_critic, actor_critic.critic_1, off_policy_runner.replay_buffer, rng);
//            lic::train_critic(actor_critic, actor_critic.critic_2, off_policy_runner.replay_buffer, rng);
            hipMemcpy(actor_critic_gpu, &actor_critic                   , sizeof(actor_critic                   ), hipMemcpyHostToDevice);
            hipMemcpy(    critic_2_gpu, &actor_critic.critic_2          , sizeof(actor_critic.critic_2          ), hipMemcpyHostToDevice);
            hipMemcpy(          rb_gpu, &off_policy_runner.replay_buffer, sizeof(off_policy_runner.replay_buffer), hipMemcpyHostToDevice);

            dim3 grid(1);
            dim3 block(1);
            hipDeviceSynchronize();
            train_critic_kernel<<<grid, block>>>(actor_critic_gpu, critic_2_gpu, rb_gpu, rng_gpu, off_policy_runner.replay_buffer.position == N_WARMUP_STEPS + 1);
            hipDeviceSynchronize();

            std::cout << "Critic 1 loss: " << std::endl;
            if(step_i % 2 == 0){
                lic::train_actor(actor_critic, off_policy_runner.replay_buffer, rng);
                lic::update_targets(actor_critic);
            }
        }
        if(step_i % 1000 == 0){
            DTYPE mean_return = lic::evaluate<DEVICE, ENVIRONMENT, decltype(actor_critic.actor), typeof(rng), ENVIRONMENT_STEP_LIMIT, true>(device, env, actor_critic.actor, 1, rng);
            std::cout << "Mean return: " << mean_return << std::endl;
        }
    }
    return 0;
}
